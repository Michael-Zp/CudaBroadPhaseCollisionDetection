#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "DataTypes.h"
#include "Constants.h"
#include "Helpers.h"


__global__ void VecAdd()
{

}



int main()
{
	hipError_t err = hipSuccess;

	Circle *circles = (Circle*)malloc(sizeof(Circle) * OBJECT_COUNT);
	

	circles[0].center.x = 0;
	circles[0].center.y = 0;
	circles[0].radius = CELLSIZE / 4;

	circles[1].center.x = 0;
	circles[1].center.y = 1;
	circles[1].radius = CELLSIZE / 4;

	circles[2].center.x = 1;
	circles[2].center.y = 0;
	circles[2].radius = CELLSIZE / 4;

	circles[3].center.x = 1;
	circles[3].center.y = 1;
	circles[3].radius = CELLSIZE / 4;



	CellIdItem *cellIds = (CellIdItem*)malloc(sizeof(CellIdItem) * MAX_ITEMS);
	ControlBitsItem *controlBits = (ControlBitsItem*)malloc(sizeof(ControlBitsItem) * MAX_ITEMS);
	


	for (int i = 0; i < OBJECT_COUNT; i++)
	{
		cellIds[i].Cells[0] = posToHash(circles[i].center);
		controlBits[i].HCellType = posToCellType(circles[i].center);
		
		glm::uvec2 coords = posToCoords(circles[i].center);

		int collisionCount = 0;

		for (int x = -1; x <= 1; x++)
		{
			for (int y = -1; y <= 1; y++)
			{
				if (x == 0 && y == 0)
				{
					continue;
				}

				GLuint currentX = coords.x + x;
				GLuint currentY = coords.y + y;

				if (collides(circles[i], coordsToGridBox(currentX, currentY)))
				{
					cellIds[i].Cells[collisionCount + 1] = posToHash(circles[i].center + glm::vec2(x * CELLSIZE, y * CELLSIZE));
					collisionCount++;
				}
			}
		}

		for (int u = collisionCount + 1; u < MAX_OBJECT_INTERSECTIONS; u++)
		{
			cellIds[i].Cells[u] = 0xffffffff;
		}
	}

	for (int i = 0; i < OBJECT_COUNT; i++)
	{
		printf("cellIds[%d] HomeCell = %d\n", i, cellIds[i].Cells[0]);
		printf("cellIds[%d] HomeCellType = %d\n", i, controlBits[i].HCellType);
		for (int u = 0; u < MAX_OBJECT_INTERSECTIONS; u++)
		{
			printf("cellIds[%d] Cells [%d] = %d\n", i, u, cellIds[i].Cells[u]);
		}
		printf("----\n");
	}

    return err;
}