#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void VecAdd(const float* A, const float* B, float* C)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	C[i] = A[i] + B[i];
}


int main()
{
	hipError_t err = hipSuccess;

	float A[3] = { 1, 2, 3 };
	float B[3] = { 6, 2, 1 };
	float C[3];

	float size = 3 * sizeof(float);

	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);
	err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	float *d_B = NULL;
	err = hipMalloc((void **)&d_B, size);
	err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	float *d_C = NULL;
	err = hipMalloc((void **)&d_C, size);

	VecAdd << <3, 1 >> > (d_A, d_B, d_C);

	err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	printf("A = (%f, %f, %f)", A[0], A[1], A[2]);
	printf("B = (%f, %f, %f)", B[0], B[1], B[2]);
	printf("C = (%f, %f, %f)", C[0], C[1], C[2]);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);


	return err;
}